#include "hip/hip_runtime.h"
#include "FlipMetricImpl.cuh"

#include <cmath>

#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

__device__ float __cuFlip_min(float x, float y) {
    return x < y ? x : y;
}

__device__ float __cuFlip_max(float x, float y) {
    return x > y ? x : y;
}

__device__ float __cuFlip_abs(float x) {
    return x < 0 ? -x : x;
}

__device__ float __cuFlip_HyAB(const Color& refPixel, const Color& testPixel) {
    float cityBlockDistanceL = fabsf(refPixel.x - testPixel.x);
    float euclideanDistanceAB = sqrtf((refPixel.y - testPixel.y) * (refPixel.y - testPixel.y) + (refPixel.z - testPixel.z) * (refPixel.z - testPixel.z));
    return cityBlockDistanceL + euclideanDistanceAB;
}

__device__ float __cuFlip_GaussSum(const float x2, const float a1, const float b1, const float a2, const float b2) {
    const float pi = float(M_PI);
    const float pi_sq = float(M_PI * M_PI);
    return a1 * sqrtf(pi / b1) * expf(-pi_sq * x2 / b1) + a2 * sqrtf(pi / b2) * expf(-pi_sq * x2 / b2);
}

__device__ float __cuFlip_sRGB2Linear(float sRGBColor) {
    if (sRGBColor <= 0.04045f) {
        return sRGBColor / 12.92f;
    } else {
        return powf((sRGBColor + 0.055f) / 1.055f, 2.4f);
    }
}

__device__ void __cuFlip_LinearRGB2XYZ(float& r, float& g, float& b) {
    const float a11 = 10135552.0f / 24577794.0f;
    const float a12 = 8788810.0f / 24577794.0f;
    const float a13 = 4435075.0f / 24577794.0f;
    const float a21 = 2613072.0f / 12288897.0f;
    const float a22 = 8788810.0f / 12288897.0f;
    const float a23 = 887015.0f / 12288897.0f;
    const float a31 = 1425312.0f / 73733382.0f;
    const float a32 = 8788810.0f / 73733382.0f;
    const float a33 = 70074185.0f / 73733382.0f;
    float vR = r;
    float vG = g;
    float vB = b;
    r = a11 * vR + a12 * vG + a13 * vB;
    g = a21 * vR + a22 * vG + a23 * vB;
    b = a31 * vR + a32 * vG + a33 * vB;
}

__global__ void __cuFlip_sRGB2YCxCz(const unsigned char* input, Color* output, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    Color reference_illuminant = {0.950428545377181f, 1.0f, 1.088900370798128f};

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        float x = __cuFlip_sRGB2Linear(input[i * 3 + 0] / 255.0f);
        float y = __cuFlip_sRGB2Linear(input[i * 3 + 1] / 255.0f);
        float z = __cuFlip_sRGB2Linear(input[i * 3 + 2] / 255.0f);

        __cuFlip_LinearRGB2XYZ(x, y, z);

        x /= reference_illuminant.x;
        y /= reference_illuminant.y;
        z /= reference_illuminant.z;

        Color& out = output[i];
        out.x = 116.0f * y - 16.0f;
        out.y = 500.0f * (x - y);
        out.z = 200.0f * (y - z);
    }
}

__global__ void __cuFlip_YCxCz2Gray(const Color* input, Color* output, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        const Color& pixel = input[i];

        float c = (pixel.x + 16.0f) / 116.0f;

        Color& out = output[i];
        out.x = c;
        out.y = c;
        out.z = 0.0f;
    }
}

__global__ void __cuFlip_YCxCz2CIELab(const Color* input, Color* output, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    Color reference_illuminant = {0.950428545377181f, 1.0f, 1.088900370798128f};

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        const Color& src = input[i];
        Color& out = output[i];
        Color YCxCz = src;

        const float Yy = (YCxCz.x + 16.0f) / 116.0f;
        const float Cx = YCxCz.y / 500.0f;
        const float Cz = YCxCz.z / 200.0f;
        out.x = Yy + Cx;
        out.y = Yy;
        out.z = Yy - Cz;
        out.x *= reference_illuminant.x;
        out.y *= reference_illuminant.y;
        out.z *= reference_illuminant.z;

        const float a11 = 3.241003232976358f;
        const float a12 = -1.537398969488785f;
        const float a13 = -0.498615881996363f;
        const float a21 = -0.969224252202516f;
        const float a22 = 1.875929983695176f;
        const float a23 = 0.041554226340085f;
        const float a31 = 0.055639419851975f;
        const float a32 = -0.204011206123910f;
        const float a33 = 1.057148977187533f;
        Color v = out;
        out.x = __cuFlip_min(__cuFlip_max(a11 * v.x + a12 * v.y + a13 * v.z, 0.0f), 1.0f);
        out.y = __cuFlip_min(__cuFlip_max(a21 * v.x + a22 * v.y + a23 * v.z, 0.0f), 1.0f);
        out.z = __cuFlip_min(__cuFlip_max(a31 * v.x + a32 * v.y + a33 * v.z, 0.0f), 1.0f);

        const float b11 = 10135552.0f / 24577794.0f;
        const float b12 = 8788810.0f / 24577794.0f;
        const float b13 = 4435075.0f / 24577794.0f;
        const float b21 = 2613072.0f / 12288897.0f;
        const float b22 = 8788810.0f / 12288897.0f;
        const float b23 = 887015.0f / 12288897.0f;
        const float b31 = 1425312.0f / 73733382.0f;
        const float b32 = 8788810.0f / 73733382.0f;
        const float b33 = 70074185.0f / 73733382.0f;
        v = out;
        out.x = __cuFlip_abs(b11 * v.x + b12 * v.y + b13 * v.z);
        out.y = __cuFlip_abs(b21 * v.x + b22 * v.y + b23 * v.z);
        out.z = __cuFlip_abs(b31 * v.x + b32 * v.y + b33 * v.z);

        Color xyz = out;
        xyz.x /= reference_illuminant.x;
        xyz.y /= reference_illuminant.y;
        xyz.z /= reference_illuminant.z;
        xyz.x = xyz.x > 0.008856 ? powf(xyz.x, 1.0f / 3.0f) : 7.787f * xyz.x + 16.0f / 116.0f;
        xyz.y = xyz.y > 0.008856 ? powf(xyz.y, 1.0f / 3.0f) : 7.787f * xyz.y + 16.0f / 116.0f;
        xyz.z = xyz.z > 0.008856 ? powf(xyz.z, 1.0f / 3.0f) : 7.787f * xyz.z + 16.0f / 116.0f;
        out.x = 116.0f * xyz.y - 16.0f;
        out.y = 500.0f * (xyz.x - xyz.y);
        out.z = 200.0f * (xyz.y - xyz.z);
    }
}

__global__ void __cuFlip_huntAdjustment(const Color* input, Color* output, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        const Color& src = input[i];
        Color& out = output[i];

        out.y = 0.01f * src.x * src.y;
        out.z = 0.01f * src.x * src.z;
    }
}

__global__ void __cuFlip_normalize(const Color* input, Color* output, Color total, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        const Color& src = input[i];
        Color& out = output[i];
        out.x = src.x / total.x;
        out.y = src.y / total.y;
        out.z = src.z / total.z;
    }
}

__global__ void __cuFlip_generateSpatialFilter(Color* output, unsigned int width, float radius, float deltaX) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    Color a1 = {1.0f, 1.0f, 34.1f};
    Color b1 = {0.0047f, 0.0053f, 0.04f };
    Color a2 = { 0.0f, 0.0f, 13.5f };
    Color b2 = { 1.0e-5f, 1.0e-5f, 0.025f };

    unsigned int pixelsCount = width * width;

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        unsigned int x = i % width;
        unsigned int y = i / width;

        float iy = (y - radius) * deltaX;
        float ix = (x - radius) * deltaX;

        float dist2 = ix * ix + iy * iy;
        output[i] = Color{__cuFlip_GaussSum(dist2, a1.x, b1.x, a2.x, b2.x), __cuFlip_GaussSum(dist2, a1.y, b1.y, a2.y, b2.y), __cuFlip_GaussSum(dist2, a1.z, b1.z, a2.z, b2.z)};
    }
}

__global__ void __cuFlip_computeColorDifference(const Color* reference, const Color* test, Color* output, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    const float gpc = 0.4f;
    const float gqc = 0.7f;
    const float gpt = 0.95f;
    const float cmax = 41.2761f;
    const float pccmax = gpc * cmax;

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        // compute difference in HyAB
        Color refPixel = reference[i];
        Color testPixel = test[i];
        float error = __cuFlip_HyAB(refPixel, testPixel);

        error = powf(error, gqc);

        // Re-map error to the [0, 1] range. Values between 0 and pccmax are mapped to the range [0, gpt],
        // while the rest are mapped to the range (gpt, 1]
        if (error < pccmax) {
            error *= gpt / pccmax;
        } else {
            error = gpt + ((error - pccmax) / (cmax - pccmax)) * (1.0f - gpt);
        }

        Color& out = output[i];
        out.x = error;
        out.y = 0.0f;
        out.z = 0.0f;
    }
}

__global__ void __cuFlip_computeFeatureDifference(const Color* edgesReference, const Color* edgesTest, const Color* pointsReference, const Color* pointsTest, Color* output, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    const float gqf = 0.5f;
    const float normalizationFactor = 1.0f / sqrtf(2.0f);
    Color p;

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        p = edgesReference[i];
        const float edgeValueRef = sqrtf(p.x * p.x + p.y * p.y);

        p = edgesTest[i];
        const float edgeValueTest = sqrtf(p.x * p.x + p.y * p.y);

        p = pointsReference[i];
        const float pointValueRef = sqrtf(p.x * p.x + p.y * p.y);

        p = pointsTest[i];
        const float pointValueTest = sqrtf(p.x * p.x + p.y * p.y);

        const float edgeDifference = __cuFlip_abs(edgeValueRef - edgeValueTest);
        const float pointDifference = __cuFlip_abs(pointValueRef - pointValueTest);

        const float featureDifference = pow(normalizationFactor * __cuFlip_max(edgeDifference, pointDifference), gqf);

        Color& out = output[i];
        out.x = featureDifference;
        out.y = 0.0f;
        out.z = 0.0f;
    }
}

__global__ void __cuFlip_computeFlipError(const Color* colorDifference, const Color* featureDifference, float* output, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        const float cdiff = colorDifference[i].x;
        const float fdiff = featureDifference[i].x;
        const float errorFLIP = std::pow(cdiff, 1.0f - fdiff);

        output[i] = errorFLIP;
    }
}

__global__ void __cuFlip_hwc2chw(const Color* input, float* output, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        output[i + pixelsCount * 0] = input[i].x;
        output[i + pixelsCount * 1] = input[i].y;
        output[i + pixelsCount * 2] = input[i].z;
    }
}

__global__ void __cuFlip_chw2hwc(const float* input, Color* output, unsigned int pixelsCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < pixelsCount; i += stride) {
        output[i].x = input[i + pixelsCount * 0];
        output[i].y = input[i + pixelsCount * 1];
        output[i].z = input[i + pixelsCount * 2];
    }
}

FlipMetricImpl::Filter::Filter(hipdnnHandle_t cudnnHandle, hipdnnTensorDescriptor_t inputDescriptor, hipdnnTensorDescriptor_t outputDescriptor, unsigned int width, unsigned int height) {
    m_hwc.resize(width * height);
    m_chw.resize(width * height * 3);

    hipdnnCreateFilterDescriptor(&m_filterDescriptor);
    hipdnnSetFilter4dDescriptor(m_filterDescriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, 1, height, width);

    hipdnnCreateConvolutionDescriptor(&m_convolutionDescriptor);
    hipdnnSetConvolution2dDescriptor(m_convolutionDescriptor, height / 2, width / 2, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    int count;
    hipdnnFindConvolutionForwardAlgorithm(cudnnHandle, inputDescriptor, m_filterDescriptor, m_convolutionDescriptor, outputDescriptor, 1, &count, &m_convolutionAlgorithm);

    hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle, inputDescriptor, m_filterDescriptor, m_convolutionDescriptor, outputDescriptor, m_convolutionAlgorithm.algo, &m_workspaceSize);

    hipMalloc(&m_workspace, m_workspaceSize);
}

FlipMetricImpl::Filter::~Filter() {
    hipdnnDestroyFilterDescriptor(m_filterDescriptor);
    hipdnnDestroyConvolutionDescriptor(m_convolutionDescriptor);
    hipFree(&m_workspace);
}

hipdnnFilterDescriptor_t FlipMetricImpl::Filter::getFilterDescriptor() {
    return m_filterDescriptor;
}

hipdnnConvolutionDescriptor_t FlipMetricImpl::Filter::getConvolutionDescriptor() {
    return m_convolutionDescriptor;
}

hipdnnConvolutionFwdAlgo_t FlipMetricImpl::Filter::getConvolutionAlgorithm() {
    return m_convolutionAlgorithm.algo;
}

void* FlipMetricImpl::Filter::getWorkspace() {
    return m_workspace;
}

size_t FlipMetricImpl::Filter::getWorkspaceSize() {
    return m_workspaceSize;
}

FlipMetricImpl::ColorVec& FlipMetricImpl::Filter::getDataHWC() {
    return m_hwc;
}

thrust::device_vector<float>& FlipMetricImpl::Filter::getDataCHW() {
    return m_chw;
}

void FlipMetricImpl::Filter::toCHW() {
    __cuFlip_hwc2chw<<<128, 4>>>(thrust::raw_pointer_cast(m_hwc.data()), thrust::raw_pointer_cast(m_chw.data()), m_hwc.size());
    hipDeviceSynchronize();
}

float FlipMetricImpl::gaussian(const float x, const float y, const float sigma) {
    return expf(-(x * x + y * y) / (2.0f * sigma * sigma));
}

void FlipMetricImpl::sRGB2YCxCz(const unsigned char* input, ColorVec& output) {
    __cuFlip_sRGB2YCxCz<<<128, 4>>>(input, thrust::raw_pointer_cast(output.data()), output.size());
}

void FlipMetricImpl::YCxCz2Gray(const ColorVec& input, ColorVec& output) {
    __cuFlip_YCxCz2Gray<<<128, 4>>>(thrust::raw_pointer_cast(input.data()), thrust::raw_pointer_cast(output.data()), output.size());
}

void FlipMetricImpl::YCxCz2CIELab(const ColorVec& input, ColorVec& output) {
    __cuFlip_YCxCz2CIELab<<<128, 4>>>(thrust::raw_pointer_cast(input.data()), thrust::raw_pointer_cast(output.data()), output.size());
}

void FlipMetricImpl::huntAdjustment(const ColorVec& input, ColorVec& output) {
    __cuFlip_huntAdjustment<<<128, 4>>>(thrust::raw_pointer_cast(input.data()), thrust::raw_pointer_cast(output.data()), output.size());
}

void FlipMetricImpl::normalize(const ColorVec& input, ColorVec& output, const Color& total) {
    __cuFlip_normalize<<<128, 4>>>(thrust::raw_pointer_cast(input.data()), thrust::raw_pointer_cast(output.data()), total, output.size());
}

void FlipMetricImpl::generateSpatialFilter(ColorVec& output, unsigned int width, float radius, float deltaX) {
    __cuFlip_generateSpatialFilter<<<128, 4>>>(thrust::raw_pointer_cast(output.data()), width, radius, deltaX);
}

void FlipMetricImpl::convolve(const std::shared_ptr<Filter>& filter, const ColorVec& input, ColorVec& output) {
    __cuFlip_hwc2chw<<<128, 4>>>(thrust::raw_pointer_cast(input.data()), thrust::raw_pointer_cast(m_chwInput.data()), input.size());
    hipDeviceSynchronize();

    float alpha = 1.0f;
    float beta = 0.0f;
    hipdnnConvolutionForward(m_cudnnHandle,
                            &alpha, m_inputDescriptor, thrust::raw_pointer_cast(m_chwInput.data()),
                            filter->getFilterDescriptor(), thrust::raw_pointer_cast(filter->getDataCHW().data()),
                            filter->getConvolutionDescriptor(), filter->getConvolutionAlgorithm(),
                            filter->getWorkspace(), filter->getWorkspaceSize(),
                            &beta, m_outputDescriptor, thrust::raw_pointer_cast(m_chwOutput.data()));
    __cuFlip_chw2hwc<<<128, 4>>>(thrust::raw_pointer_cast(m_chwOutput.data()), thrust::raw_pointer_cast(output.data()), output.size());
    hipDeviceSynchronize();
}

void FlipMetricImpl::computeColorDifference(const ColorVec& reference, const ColorVec& test, ColorVec& output) {
    __cuFlip_computeColorDifference<<<128, 4>>>(thrust::raw_pointer_cast(reference.data()), thrust::raw_pointer_cast(test.data()), thrust::raw_pointer_cast(output.data()), output.size());
}

void FlipMetricImpl::computeFeatureDifference(const ColorVec& edgesReference, const ColorVec& edgesTest, const ColorVec& pointsReference, const ColorVec& pointsTest, ColorVec& output) {
    __cuFlip_computeFeatureDifference<<<128, 4>>>(thrust::raw_pointer_cast(edgesReference.data()), thrust::raw_pointer_cast(edgesTest.data()), thrust::raw_pointer_cast(pointsReference.data()), thrust::raw_pointer_cast(pointsTest.data()), thrust::raw_pointer_cast(output.data()), output.size());
}

void FlipMetricImpl::computeFlipError(const ColorVec& colorDiff, const ColorVec& featureDiff, thrust::device_vector<float>& output) {
    __cuFlip_computeFlipError<<<128, 4>>>(thrust::raw_pointer_cast(colorDiff.data()), thrust::raw_pointer_cast(featureDiff.data()), thrust::raw_pointer_cast(output.data()), output.size());
}

void FlipMetricImpl::createColorFilter() {
    const float deltaX = 1.0f / m_ppd;
    const float pi_sq = float(M_PI * M_PI);
    // constants for Gaussians -- see paper for details.
    Color b1 = {0.0047f, 0.0053f, 0.04f };
    Color b2 = { 1.0e-5f, 1.0e-5f, 0.025f };

    float maxScaleParameter = std::max(std::max(std::max(b1.x, b1.y), std::max(b1.z, b2.x)), std::max(b2.y, b2.z));
    int radius = int(std::ceil(3.0f * sqrtf(maxScaleParameter / (2.0f * pi_sq)) * m_ppd));

    unsigned int width = 2 * radius + 1;
    m_colorFilter = std::make_shared<Filter>(m_cudnnHandle, m_inputDescriptor, m_outputDescriptor, width, width);

    ColorVec& filterData = m_colorFilter->getDataHWC();

    generateSpatialFilter(filterData, width, radius, deltaX);
    hipDeviceSynchronize();

    Color totalFilterColor = thrust::reduce(filterData.begin(), filterData.end(), Color{0.0f, 0.0f, 0.0f});
    normalize(filterData, filterData, totalFilterColor);
    hipDeviceSynchronize();

    m_colorFilter->toCHW();
}

void FlipMetricImpl::createDetectionFilter(ColorVec& output, float stdDev, float radius, int width, bool pointDetector) {
    float weightX, weightY;
    float negativeWeightsSumX = 0.0f;
    float positiveWeightsSumX = 0.0f;
    float negativeWeightsSumY = 0.0f;
    float positiveWeightsSumY = 0.0f;

    for (int y = 0; y < width; y++) {
        int yy = y - radius;
        for (int x = 0; x < width; x++) {
            int xx = x - radius;
            float G = gaussian(float(xx), float(yy), stdDev);
            if (pointDetector) {
                weightX = (float(xx) * float(xx) / (stdDev * stdDev) - 1.0f) * G;
                weightY = (float(yy) * float(yy) / (stdDev * stdDev) - 1.0f) * G;
            } else {
                weightX = -float(xx) * G;
                weightY = -float(yy) * G;
            }

            output[y * width + x] = Color{weightX, weightY, 0.0f};

            if (weightX > 0.0f) {
                positiveWeightsSumX += weightX;
            } else {
                negativeWeightsSumX += -weightX;
            }

            if (weightY > 0.0f) {
                positiveWeightsSumY += weightY;
            } else {
                negativeWeightsSumY += -weightY;
            }
        }
    }

    // Normalize positive weights to sum to 1 and negative weights to sum to -1
    for (int y = 0; y < width; y++) {
        for (int x = 0; x < width; x++) {
            Color p = output[y * width + x];
            output[y * width + x] = Color{p.x / (p.x > 0.0f ? positiveWeightsSumX : negativeWeightsSumX), p.y / (p.y > 0.0f ? positiveWeightsSumY : negativeWeightsSumY), 0.0f};
        }
    }
}

void FlipMetricImpl::createDetectionFilters() {
    const float gw = 0.082f;
    const float stdDev = 0.5f * gw * m_ppd;
    const int radius = int(std::ceil(3.0f * stdDev));

    unsigned int width = 2 * radius + 1;

    m_edgesFilter = std::make_shared<Filter>(m_cudnnHandle, m_inputDescriptor, m_outputDescriptor, width, width);
    m_pointsFilter = std::make_shared<Filter>(m_cudnnHandle, m_inputDescriptor, m_outputDescriptor, width, width);

    createDetectionFilter(m_edgesFilter->getDataHWC(), stdDev, radius, width, false);
    m_edgesFilter->toCHW();

    createDetectionFilter(m_pointsFilter->getDataHWC(), stdDev, radius, width, true);
    m_pointsFilter->toCHW();
}

void FlipMetricImpl::preprocess(ColorVec& image, ColorVec& imageGray) {
    YCxCz2Gray(image, imageGray);
    hipDeviceSynchronize();

    convolve(m_colorFilter, image, image);
    hipDeviceSynchronize();

    YCxCz2CIELab(image, image);
    hipDeviceSynchronize();

    huntAdjustment(image, image);
    hipDeviceSynchronize();
}

FlipMetricImpl::FlipMetricImpl(const unsigned char* image, unsigned int width, unsigned int height, float ppd) {
    m_ppd = ppd;
    m_imageWidth = width;
    m_imageHeight = height;

    hipdnnCreate(&m_cudnnHandle);

    hipdnnCreateTensorDescriptor(&m_inputDescriptor);
    hipdnnSetTensor4dDescriptor(m_inputDescriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 3, m_imageHeight, m_imageWidth);

    hipdnnCreateTensorDescriptor(&m_outputDescriptor);
    hipdnnSetTensor4dDescriptor(m_outputDescriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 3, m_imageHeight, m_imageWidth);

    createColorFilter();
    createDetectionFilters();

    m_referencePixels.resize(m_imageWidth * m_imageHeight);
    m_referenceGrayPixels.resize(m_imageWidth * m_imageHeight);
    m_testPixels.resize(m_imageWidth * m_imageHeight);
    m_testGrayPixels.resize(m_imageWidth * m_imageHeight);
    m_colorDifference.resize(m_imageWidth * m_imageHeight);
    m_featureDifference.resize(m_imageWidth * m_imageHeight);

    m_edgesReference.resize(m_imageWidth * m_imageHeight);
    m_edgesTest.resize(m_imageWidth * m_imageHeight);
    m_pointsReference.resize(m_imageWidth * m_imageHeight);
    m_pointsTest.resize(m_imageWidth * m_imageHeight);

    m_chwInput.resize(m_imageWidth * m_imageHeight * 3);
    m_chwOutput.resize(m_imageWidth * m_imageHeight * 3);

    m_flip.resize(m_imageWidth * m_imageHeight);
    m_histogram.resize(100);
    m_histogramSeq.resize(m_histogram.size());
    thrust::sequence(m_histogramSeq.begin(), m_histogramSeq.end(), 0.0f, 1.0f / m_histogramSeq.size());

    thrust::device_vector<unsigned char> imageDevice(m_imageWidth * m_imageHeight * 3);
    thrust::copy(image, image + imageDevice.size(), imageDevice.begin());
    sRGB2YCxCz(thrust::raw_pointer_cast(imageDevice.data()), m_referencePixels);
    hipDeviceSynchronize();

    preprocess(m_referencePixels, m_referenceGrayPixels);
}

FlipMetricImpl::~FlipMetricImpl() {
    hipdnnDestroyTensorDescriptor(m_inputDescriptor);
    hipdnnDestroyTensorDescriptor(m_outputDescriptor);
    hipdnnDestroy(m_cudnnHandle);
}

float getWeightedPercentile(const thrust::device_vector<float> histogram, const double percent) {
    double weight;
    double weightedValue;
    double bucketStep = 1.0f / histogram.size();
    double sumWeightedDataValue = 0.0;
    for (size_t bucketId = 0; bucketId < histogram.size(); bucketId++)
    {
        weight = (bucketId + 0.5) * bucketStep;
        weightedValue = histogram[bucketId] * weight;
        sumWeightedDataValue += weightedValue;
    }

    double sum = 0;
    size_t weightedMedianIndex = 0;
    for (size_t bucketId = 0; bucketId < histogram.size(); bucketId++)
    {
        weight = (bucketId + 0.5) * bucketStep;
        weightedValue = histogram[bucketId] * weight;
        weightedMedianIndex = bucketId;
        if (sum + weightedValue > percent * sumWeightedDataValue)
            break;
        sum += weightedValue;
    }

    weight = (weightedMedianIndex + 0.5) * bucketStep;
    weightedValue = histogram[weightedMedianIndex] * weight;
    double discrepancy = percent * sumWeightedDataValue - sum;
    double linearWeight = discrepancy / weightedValue; // in [0,1]
    double percentile = (weightedMedianIndex + linearWeight) * bucketStep;
    return percentile;
}

float FlipMetricImpl::compareDevice(const unsigned char* image) {
    sRGB2YCxCz(image, m_testPixels);
    hipDeviceSynchronize();

    preprocess(m_testPixels, m_testGrayPixels);

    computeColorDifference(m_referencePixels, m_testPixels, m_colorDifference);
    convolve(m_edgesFilter, m_referenceGrayPixels, m_edgesReference);
    convolve(m_edgesFilter, m_testGrayPixels, m_edgesTest);
    convolve(m_pointsFilter, m_referenceGrayPixels, m_pointsReference);
    convolve(m_pointsFilter, m_testGrayPixels, m_pointsTest);
    hipDeviceSynchronize();

    computeFeatureDifference(m_edgesReference, m_edgesTest, m_pointsReference, m_pointsTest, m_featureDifference);
    hipDeviceSynchronize();

    computeFlipError(m_colorDifference, m_featureDifference, m_flip);
    hipDeviceSynchronize();

    thrust::sort(m_flip.begin(), m_flip.end());
    thrust::upper_bound(m_flip.begin(), m_flip.end(), m_histogramSeq.begin(), m_histogramSeq.end(), m_histogram.begin());
    thrust::adjacent_difference(m_histogram.begin(), m_histogram.end(), m_histogram.begin());

    return getWeightedPercentile(m_histogram, 0.5f);
}

float FlipMetricImpl::compareHost(const unsigned char* image) {
    thrust::device_vector<unsigned char> imageDevice(m_imageWidth * m_imageHeight * 3);
    thrust::copy(image, image + imageDevice.size(), imageDevice.begin());
    return compareDevice(thrust::raw_pointer_cast(imageDevice.data()));
}
